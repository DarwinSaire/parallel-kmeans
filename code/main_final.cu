
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h> 
#include <sys/types.h>
#include <math.h>
#include <errno.h>

__host__ __device__ float func_distance(int numdim, float *dim_pointX, int id_point, float *dim_pointY, int id_cluster ){
    int i;
    float ans = 0.0;
    for( i = 0; i < numdim; i++ )
        ans += ( dim_pointX[ id_point * numdim + i ] - dim_pointY[ id_cluster * numdim + i ] ) * ( dim_pointX[ id_point * numdim + i ]- dim_pointY[ id_cluster * numdim + i ] );
    return ans;
}

__host__ __device__ int func_find_nearest_cluster(int numClusters, int numdim, float  *dim_point, float *clusters, int id_point ){
    int i, nearest_cluster = 0;
    float dist;
    float min_dist = func_distance( numdim, dim_point, id_point, clusters, 0 );

    for( i = 1; i < numClusters; i++ ){
        dist = func_distance( numdim, dim_point, id_point, clusters, i );
        if( dist < min_dist ){
            min_dist = dist;
            nearest_cluster  = i;
        }
    }
    return nearest_cluster;
}

__global__ void Func_kmeans_point(float *dim_point, int numdim, int num_point, int numClusters, int *belong_point, float *clusters, float *newClusters, int *newClusterSize ){

    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if( i > num_point ) return;

    int nearest_cluster;
    if( i < num_point ) {
	nearest_cluster = func_find_nearest_cluster( numClusters, numdim, dim_point, clusters, i );
	belong_point[ i ] = nearest_cluster;
	atomicAdd( &newClusterSize[ nearest_cluster ], 1 );
	//newClusterSize[ nearest_cluster ] += 1;
	for( int j = 0; j < numdim; j++ ){
		float num = dim_point[ i * numdim + j ];
		atomicAdd( &newClusters[ nearest_cluster * numdim + j ], num );
	}
		//newClusters[ nearest_cluster * numdim + j ] += dim_point[ i * numdim + j ] ;
    } 
}

int func_kmeans(float *dim_point, int numdim, int num_point, int numClusters, int *belong_point, float *clusters, int numthreads ){
    int i, j, counter = 0;
    /*numero de objetos que tem cada cluster*/
    int *newClusterSize;
    /*dimensões da nova mudanca dos clusters*/
    float *newClusters;

    //dim_point, clusters, newClusters
    
    newClusterSize = ( int* )calloc( numClusters, sizeof( int ) );
    newClusters = ( float* )calloc( numClusters * numdim, sizeof( float ) );

    for( i = 0; i < num_point; i++ ) belong_point[ i ] = -1;

    /*for( i = 1; i < numClusters; i++ )
        newClusters[ i ] = newClusters[ i - 1 ] + numdim;*/

    float * device_dim_point, *device_clusters, *device_newclusters;
    int *device_belong_point, *device_newclustersize;

    int size_device_dimpoint = num_point * numdim * sizeof( float );
    int size_device_clusters = numClusters * numdim * sizeof( float );
    int size_device_newclusters = numClusters * numdim * sizeof( float );
    int size_device_belong = num_point * sizeof( int );
    int size_device_newclusersize = numClusters * sizeof( int );

    hipMalloc( ( void** ) &device_dim_point, size_device_dimpoint );
    hipMalloc( ( void** ) &device_clusters, size_device_clusters );
    hipMalloc( ( void** ) &device_newclusters, size_device_newclusters );
    hipMalloc( ( void** ) &device_belong_point, size_device_belong );
    hipMalloc( ( void** ) &device_newclustersize, size_device_newclusersize );

    hipMemset(device_newclusters,0,size_device_newclusters);

    hipMemcpy( device_dim_point, dim_point, size_device_dimpoint, hipMemcpyHostToDevice );

    int threads_block = 256;
    dim3 DimGrid( ( num_point - 1 ) / threads_block + 1, 1, 1 );
    dim3 DimBlock( threads_block, 1, 1 );

    do {
	/*selecionar o cluster mais próximo para cada ponto e sumar as dimensões em newCluster
	  para obtem um novo cluster*/

	hipMemcpy( device_clusters, clusters, size_device_clusters, hipMemcpyHostToDevice );
	hipMemcpy( device_newclusters, newClusters, size_device_newclusters, hipMemcpyHostToDevice );
	hipMemcpy( device_belong_point, belong_point, size_device_belong, hipMemcpyHostToDevice );
	hipMemcpy( device_newclustersize, newClusterSize, size_device_newclusersize, hipMemcpyHostToDevice );

	//(float * dim_point, int numdim, int num_point, int numClusters, int *belong_point, float *clusters, int *newClusterSize, int *newClusters )

	Func_kmeans_point<<< DimGrid, DimBlock >>>( device_dim_point, numdim, num_point, numClusters, device_belong_point, device_clusters, device_newclusters, device_newclustersize );

	hipMemcpy( clusters, device_clusters, size_device_clusters, hipMemcpyDeviceToHost );
	hipMemcpy( newClusters ,device_newclusters, size_device_newclusters, hipMemcpyDeviceToHost );
	hipMemcpy( belong_point, device_belong_point, size_device_belong, hipMemcpyDeviceToHost );
	hipMemcpy( newClusterSize, device_newclustersize, size_device_newclusersize, hipMemcpyDeviceToHost );

        /*for( i = 0; i < num_point; i++ ){
            nearest_cluster = func_find_nearest_cluster( numClusters, numdim, dim_point[ i ], clusters);
            belong_point[ i ] = nearest_cluster;
            newClusterSize[ nearest_cluster ]++;
            for( j = 0; j < numdim; j++ )
                newClusters[ nearest_cluster ][ j ] += dim_point[ i ][ j ];
        }*/

	/*obter novos clusters( medía ) e substituir os clusters antigos para os novos clusters*/
        for( i = 0; i < numClusters; i++ ){
            for( j = 0; j < numdim; j++ ){
                if( newClusterSize[ i ] > 0 )
                    clusters[ i * numdim + j ] = newClusters[ i  * numdim + j ] / newClusterSize[ i ];
                newClusters[ i * numdim + j ] = 0.0;
            }
            newClusterSize[ i ] = 0;
        }
    } while( counter++ < 1000 );

    //for( i = 0; i < num_point; i++ )
	//printf( "%d %d\n", i, belong_point[ i ] );

    //free(newClusters[0]);
    free(newClusters);
    free(newClusterSize);

    return 1;
}

/* belong_point: cluster ao qual pertence cada ponto depois de executar o kmeans
 * numClusters: cantidade de cluster
 * filename: arquivo contendo as dimensões
 * numdim: numero de dimensões que tem cada ponto
 * num_point: cantidade de ponto
 * dim_point: contém as dimensões de cada ponto, [num_point][numdim]
 * clusters: conteém as dimensões de cada cluster [numClusters][numdim]
 */

int main( int argc, char **argv ) {
	clock_t start, end;
	
    int i, j, numClusters = 0, numdim = 0, num_point = 0, *belong_point, numthreads;    
    float *dim_point, *clusters; //**dim_point, **clusters;
    //double timing, io_timing, clustering_timing;
    
    scanf( "%d %d %d %d", &num_point, &numdim, &numClusters, &numthreads );
    //printf( "%d %d %d\n", num_point, numClusters, numdim );
    
    dim_point = ( float* ) malloc( num_point * numdim * sizeof( float ) );
	
    if( numClusters <= 0 ) {
	printf( "o numero de clusters deve ser maior do que 1.\n" );
	exit( -1 );
    }
    
    for( i = 0; i < num_point; i++ ){
		//printf( "read num_point: %d\n", i );
		for( j = 0; j < numdim; j++ )
			scanf( "%f", &dim_point[ i * numdim + j ] );
	}

    clusters = ( float* ) malloc( numClusters * numdim * sizeof( float ) );
    
    /*for( i = 1; i < numClusters; i++ )
        clusters[ i ] = clusters[ i - 1 ] + numdim;*/

    for( i = 0; i < numClusters; i++ )
       for( j = 0; j < numdim; j++ )
          clusters[ i * numdim + j ] = dim_point[ i * numdim + j ];

    belong_point = (int*) malloc( num_point * sizeof( int ) );
    
    start = clock(); 

	func_kmeans( dim_point, numdim, num_point, numClusters, belong_point, clusters, numthreads );
    //func_kmeans( dim_point, numdim, num_point, numClusters, belong_point, clusters );
    
    end = clock();

    printf("time: %f seg\n", ((double)( end - start ) / CLOCKS_PER_SEC ));

    //free( dim_point[ 0 ] );
    free( dim_point );    

    free( belong_point );
    //free( clusters[ 0 ] );
    free( clusters );
    
    return 0;
}

